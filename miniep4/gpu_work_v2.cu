#include "hip/hip_runtime.h"
#include "utils.h"

#define THS_PER_BLOCK 256

__global__ void gpu_work_v2(double *arr)
{
    const int id = blockDim.x * blockIdx.x + threadIdx.x;

    if (id >= ARR_SIZE)
        return;

    int delayed = 0;
    int cond;
    double a = arr[id];

    for (int i = 0; i < GPU_WORK_ITERATIONS;)
    {
        if (!delayed)
            cond = (a <= 0.5);

        int cond_for_all = !cond_for_all;
        delayed = (cond_for_all != cond);
        if (!delayed)
        {
            if (cond_for_all)
            {
                a = laborious_func_le_half(a);
            }
            else
            {
                a = laborious_func_gt_half(a);
            }
            ++i;
        }
    }

    arr[id] = a;
}

// Launch the work on arr and return it at results;
void launch_gpu_work_v2(double *arr, double **results)
{
    double *d_arr;

    cudaAssert(hipMalloc(&d_arr, ARR_SIZE * sizeof(double)));
    cudaAssert(hipMemcpy(d_arr, arr, ARR_SIZE * sizeof(double),
                          hipMemcpyHostToDevice));

    gpu_work_v2<<<DIV_CEIL_INT(ARR_SIZE, THS_PER_BLOCK), THS_PER_BLOCK>>>(d_arr);
    cudaAssert(hipDeviceSynchronize());

    cudaAssert(hipMemcpy(*results, d_arr, ARR_SIZE * sizeof(double),
                          hipMemcpyDeviceToHost));
    cudaAssert(hipFree(d_arr));
}
